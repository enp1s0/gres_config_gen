
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sys/sysinfo.h>
#include <argp.h>

const std::string default_name = "gpu";
static const char doc[] = "SLURM GRES config file generator";
static char args_doc[]  = "";

struct arguments {
	std::string name;
	std::string header_filename;
};

static struct argp_option options[] = {
	{"header", 'h',  "FILENAME", 0, "filename of output header"},
	{"name"  , 'n',  "NAME"    , 0, "device name"},
	{0}
};

static error_t parse_opt(
		int key,
		char* arg,
		struct argp_state *state
		) {
	auto args = reinterpret_cast<arguments*>(state->input);

	switch (key) {
	case 'h':
		args->header_filename = arg;
		break;
	case 'n':
		args->name = arg;
		break;
	default:
		return ARGP_ERR_UNKNOWN;
	}
	return 0;
}

static struct argp argp = {options, parse_opt, args_doc, doc};

int main(int argc, char** argv) {
	arguments arguments;
	arguments.name = default_name;
	arguments.header_filename = "";
	argp_parse(&argp, argc, argv, 0, 0, &arguments);

	const auto nprocs = get_nprocs();

	int ngpus;
	hipGetDeviceCount(&ngpus);

	// Output
	if (arguments.header_filename != "") {
		std::ifstream ifs(arguments.name);
		if (!ifs) {
			std::fprintf(stderr, "[ERROR] No such file : %s\n", arguments.header_filename.c_str());
			return 1;
		}
		std::cout << ifs.rdbuf();
		ifs.close();
	}

	for (int gpu = 0; gpu < ngpus; gpu++) {
		std::printf("Name=%-10s File=/dev/nvidia%d CPUs=%d-%d\n",
				arguments.name.c_str(),
				gpu,
				(gpu * nprocs / ngpus),
				((gpu + 1) * nprocs / ngpus)
				);
	}
}
